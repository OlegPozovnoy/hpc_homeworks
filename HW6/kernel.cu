#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/opencv.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgcodecs.hpp>
//#include <opencv2/highgui.hpp>

using namespace std;
using namespace cv;

void task1(const char* filename_in, const char* filename_out, double* filter);
void task2(const char* filename_in, const char* filename_out, int window_size_h, int window_size_w);
void task3(const char* filename_in, const char* filename_out);

const int filter_size = 5;


__global__ void task1CUDA(uchar* src, int* d_params, double* filter) {
	int rows = d_params[0];
	int cols = d_params[1];
	int channels = d_params[2];

	int index = blockDim.x * blockIdx.x + threadIdx.x;

	double s = 0;
	for (int k = 0; k < filter_size * filter_size; k++)
		s += filter[k];

	int ch = index % channels;
	index = (index - ch) / channels;
	int row = (index) % rows;
	int col = (index) / rows;

	if (col < cols) {
			double sum = 0;
			for (int k = -2; k <= 2; k++)
				for (int l = -2; l <= 2; l++) {
					int srcRow = (row + k) < 0 ? 0 : row + k;
					srcRow = srcRow >= rows ? rows - 1 : srcRow;

					int srcCol = (col + l) < 0 ? 0 : col + l;
					srcCol = (srcCol >= cols) ? cols - 1 : srcCol;

					sum += (double)(filter[k + 2 + 5 * (l + 2)] * src[(srcRow * cols + srcCol) * channels + ch]) / s;
				}
			__syncthreads();
			src[(row * cols + col) * channels + ch] = (int)sum;
	}
}


__global__ void task2CUDA(uchar* src, int* d_params) {
	int rows = d_params[0];
	int cols = d_params[1];
	int channels = d_params[2];

	int window_h = d_params[3];
	int window_w = d_params[4];

	int index = blockDim.x * blockIdx.x + threadIdx.x;

	int ch = index % channels;
	int row = ((index - ch) / channels) % rows;
	int col = ((index - ch) / channels) / rows;

	if (col < cols) {
		//double sum = 0;
		size_t size = (2 * window_h + 1) * (2 * window_w + 1);
		uchar* values = (uchar*)malloc(size * sizeof(uchar));
		int count = 0;
		for (int k = -window_w; k <= window_w; k++)
			for (int l = -window_h; l <= window_h; l++) {
				int srcRow = (row + k) < 0 ? 0 : row + k;
				srcRow = srcRow >= rows ? rows - 1 : srcRow;

				int srcCol = (col + l) < 0 ? 0 : col + l;
				srcCol = (srcCol >= cols) ? cols - 1 : srcCol;

				values[count] = (src[(srcRow * cols + srcCol) * channels + ch]);
				count++;
			}

		for (int i = 0; i < size; i++) {
			for (int j = 0; j < size; j++) {
				for (int k = j+1; k < size; k++) {
					if (values[k] < values[j]) {
						uchar tmp = values[k];
						values[k] = values[j];
						values[j] = tmp;
					}
				}
			}
		}
		__syncthreads();
		src[(row * cols + col) * channels + ch] = values[size/2];
		free(values);
	}
}


__global__ void task3DumbVeresion(int* histohram, uchar* src, int* d_params) {
	int rows = d_params[0];
	int cols = d_params[1];
	int channels = d_params[2];

	int index = blockDim.x * blockIdx.x + threadIdx.x;

	int ch = index % channels;
	int row = ((index - ch) / channels) % rows;
	int col = ((index - ch) / channels) / rows;

	if (col < cols) {
		uchar val = (src[(row * cols + col) * channels + ch]);
		atomicAdd(&(histohram[256 * ch + val]), 1);
	}
}


int main(int argc, char** argv)
{
	const char* filename_in = "lions.jpg";
	const char* filename_out1_1 = "lions_out1_1.jpg";
	const char* filename_out1_2 = "lions_out1_2.jpg";
	const char* filename_out2 = "lions_median_filter.jpg";
	const char* filename_out3 = "histohram.txt";

	double h_filter1[filter_size * filter_size] =
	{1, 1, 1, 1, 1,
	1, 1, 1, 1, 1,
	1, 1, 1, 1, 1,
	1, 1, 1, 1, 1,
	1, 1, 1, 1, 1};

	double h_filter2[filter_size * filter_size] =
	{ 1, 1, 1, 1, 1,
	1, 0, 0, 0, 1,
	1, 0, 0, 0, 1,
	1, 0, 0, 0, 1,
	1, 1, 1, 1, 1 };

	task1(filename_in, filename_out1_1, h_filter1);
	task1(filename_in, filename_out1_2, h_filter2);

	task2(filename_in, filename_out2, 2, 2);

	task3(filename_in, filename_out3);

	return 0;
}


void task1(const char* filename_in, const char* filename_out, double* h_filter) {

	Mat src = imread(samples::findFile(filename_in), IMREAD_COLOR);

	double* d_filter1;
	uchar* d_source;

	int h_params[3] = { src.rows , src.cols, src.channels() };
	int* d_params;
	hipMalloc(&d_params, 3 * sizeof(int));
	hipMemcpy(d_params, h_params, 3 * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&d_source, src.rows * src.cols * src.channels() * sizeof(uchar));
	hipMalloc(&d_filter1, filter_size * filter_size * sizeof(double));

	hipMemcpy(d_filter1, h_filter, filter_size * filter_size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_source, src.data, src.rows * src.cols * src.channels() * sizeof(uchar), hipMemcpyHostToDevice);

	printf("\nStartCUDA");
	int blocks = src.rows * src.cols * src.channels() / 1024 + 1;
	task1CUDA << <blocks, 1024 >> > (d_source, d_params, d_filter1);
	printf("\nEndCUDA");

	uchar* destbuffer = (uchar*)malloc(src.rows * src.cols * src.channels() * sizeof(uchar));

	hipMemcpy(destbuffer, d_source, src.rows * src.cols * src.channels() * sizeof(uchar), hipMemcpyDeviceToHost);
	for (int i = 0; i < src.rows * src.cols * src.channels(); i++) {
		src.data[i] = destbuffer[i];
	}

	imwrite(filename_out, src);

	hipDeviceSynchronize();
	hipFree(d_filter1);
	hipFree(d_source);
	hipFree(d_params);
	free(destbuffer);
}


void task2(const char* filename_in, const char* filename_out, int window_size_h, int window_size_w) {

	Mat src = imread(samples::findFile(filename_in), IMREAD_COLOR);
	uchar* d_source;

	int h_params[5] = { src.rows , src.cols, src.channels(), window_size_h, window_size_w };
	int* d_params;
	hipMalloc(&d_params, 5 * sizeof(int));
	hipMemcpy(d_params, h_params, 5 * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&d_source, src.rows * src.cols * src.channels() * sizeof(uchar));
	hipMemcpy(d_source, src.data, src.rows * src.cols * src.channels() * sizeof(uchar), hipMemcpyHostToDevice);

	printf("\nStartCUDA");
	int blocks = src.rows * src.cols * src.channels() / 1024 + 1;
	task2CUDA << <blocks, 1024 >> > (d_source, d_params);
	printf("\nEndCUDA");

	uchar* destbuffer = (uchar*)malloc(src.rows * src.cols * src.channels() * sizeof(uchar));
	printf("\ncreateBuffer");
	hipMemcpy(destbuffer, d_source, src.rows * src.cols * src.channels() * sizeof(uchar), hipMemcpyDeviceToHost);
	for (int i = 0; i < src.rows * src.cols * src.channels(); i++) {
		src.data[i] = destbuffer[i];
	}
	printf("\ncopyResult");
	imwrite(filename_out, src);

	hipDeviceSynchronize();
	hipFree(d_source);
	hipFree(d_params);
	free(destbuffer);
}


void task3(const char* filename_in, const char* filename_out) {

	Mat src = imread(samples::findFile(filename_in), IMREAD_COLOR);
	uchar* d_source;

	int h_params[3] = { src.rows , src.cols, src.channels()};
	int* d_params;
	hipMalloc(&d_params, 3 * sizeof(int));
	hipMemcpy(d_params, h_params, 3 * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&d_source, src.rows * src.cols * src.channels() * sizeof(uchar));
	hipMemcpy(d_source, src.data, src.rows * src.cols * src.channels() * sizeof(uchar), hipMemcpyHostToDevice);

	int* h_histohram = (int*)calloc( src.channels()*256, sizeof(int));
	int* d_histohram;

	hipMalloc(&d_histohram, 256 * src.channels() * sizeof(int));
	hipMemcpy(d_histohram, h_histohram, 256 * src.channels() * sizeof(int), hipMemcpyHostToDevice);

	printf("\nStartCUDA");
	int blocks = src.rows * src.cols * src.channels() / 1024 + 1;
	task3DumbVeresion << <blocks, 1024 >> > (d_histohram, d_source, d_params);
	printf("\nEndCUDA");

	hipMemcpy(h_histohram, d_histohram, 256 * src.channels() * sizeof(int), hipMemcpyDeviceToHost);
	printf("\ncopyResult");

	int sum = 0;
	FILE* f = fopen(filename_out, "w");
	for (int c = 0; c < src.channels(); c++)
		for (int i = 0; i < 256; i++) {
			sum += h_histohram[c * 256 + i];
			fprintf(f, "\nchannel: %d intensity: %d count: %d", c, i, h_histohram[c * 256 + i]);
		}
	fclose(f);

	printf("compare: %d %d", sum, src.rows * src.cols * src.channels());

	hipDeviceSynchronize();
	hipFree(d_source);
	hipFree(d_params);
	hipFree(d_histohram);
	free(h_histohram);
}